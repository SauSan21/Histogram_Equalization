#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <array>
#include <time.h>

#include <hip/hip_runtime.h>
#include "image.h"

#define MAX_INTENSITY 255

#define CHECK(call)                                                       \
{                                                                         \
   const hipError_t error = call;                                        \
   if (error != hipSuccess)                                              \
   {                                                                      \
      printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
      printf("code: %d, reason: %s\n", error, hipGetErrorString(error)); \
      exit(1);                                                            \
   }                                                                      \
}

__global__ 
void calculate_histogram(int *histogram, png_byte *image, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        atomicAdd(&histogram[image[i]], 1);
    }
}

__global__ 
void compute_cdf(int *histogram, int *cdf, int length) {
    int sum = 0;
    for (int i = 0; i < length; i++) {
        sum += histogram[i];
        cdf[i] = sum;
    }
}

__global__ 
void normalize_cdf(int *cdf, int size, int min_cdf) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i <= MAX_INTENSITY) {
        cdf[i] = ((cdf[i] - min_cdf) * MAX_INTENSITY) / (size - min_cdf);
    }
}

__global__ 
void equalize(png_byte *image, int *cdf, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        image[i] = cdf[image[i]];
    }
}

int main(int argc, char *argv[]) {
    // ... (load image into "image" array and initialize "histogram" and "cdf" arrays) ...
    
    Image img = {0};
    if (argc < 2) {
        printf("Usage: %s <image.png>\n", argv[0]);
        return 1;
    }

    char *input_file = argv[1];
    read_png_file(input_file, PNG_COLOR_TYPE_GRAY, &img);
    png_byte *image = img.data[0];
    int size = img.width * img.height;
    int histogram[MAX_INTENSITY + 1] = {0};
    int cdf[MAX_INTENSITY + 1] = {0};
    
    png_byte *d_image;
    int *d_cdf;
    int *d_histogram;
    CHECK(hipMalloc(&d_histogram, (MAX_INTENSITY + 1) * sizeof(int)));
    CHECK(hipMalloc(&d_image, size * sizeof(png_byte)));
    CHECK(hipMalloc(&d_cdf, (MAX_INTENSITY + 1) * sizeof(int)));

    CHECK(hipMemcpy(d_histogram, histogram, (MAX_INTENSITY + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_image, image, size * sizeof(png_byte), hipMemcpyHostToDevice));

    calculate_histogram<<<(size + 255) / 256, 256>>>(d_histogram, d_image, size);

    CHECK(hipMemcpy(histogram, d_histogram, (MAX_INTENSITY + 1) * sizeof(int), hipMemcpyDeviceToHost));

    int threadsPerBlock = 256;
    int blocksPerGrid = (MAX_INTENSITY + threadsPerBlock - 1) / threadsPerBlock;
    compute_cdf<<<blocksPerGrid, threadsPerBlock>>>(d_histogram, d_cdf, MAX_INTENSITY + 1);

    CHECK(hipMemcpy(cdf, d_cdf, (MAX_INTENSITY + 1) * sizeof(int), hipMemcpyDeviceToHost));

    int min_cdf = cdf[0]; // Assuming cdf[0] is the minimum value in the CDF
    normalize_cdf<<<(MAX_INTENSITY + 255) / 256, 256>>>(d_cdf, size, min_cdf);

    equalize<<<(size + 255) / 256, 256>>>(d_image, d_cdf, size);

    CHECK(hipMemcpy(image, d_image, size * sizeof(png_byte), hipMemcpyDeviceToHost));



    CHECK(hipFree(d_histogram));
    CHECK(hipFree(d_image));
    CHECK(hipFree(d_cdf));


    // ... (save image to disk) ...

    char filename[50];
    sprintf(filename, "equalizer%d.png");

    // Write the equalized image to a new file each time
    write_png_file(filename, &img);

    CHECK(hipDeviceReset());

    return 0;

    
}
