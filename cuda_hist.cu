#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <array>
#include <time.h>

#include <hip/hip_runtime.h>
#include "image.h"

#define MAX_INTENSITY 255

#define CHECK(call)                                                       \
{                                                                         \
   const hipError_t error = call;                                        \
   if (error != hipSuccess)                                              \
   {                                                                      \
      printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
      printf("code: %d, reason: %s\n", error, hipGetErrorString(error)); \
      exit(1);                                                            \
   }                                                                      \
}


__global__ 
void calculate_histogram(int *histogram, png_byte *image, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        atomicAdd(&histogram[image[i]], 1);
    }
}

// __global__ 
// void exclusive_scan(int *input, png_byte *output, int length) {
//     extern __shared__ int temp[];  // allocated on invocation
//     int thid = threadIdx.x + blockIdx.x * blockDim.x;
//     int offset = 1;

//     if (thid < length) {
//         temp[thid] = input[thid]; // load input into shared memory
//     }

//     for (int d = length>>1; d > 0; d >>= 1) { // build sum in place up the tree
//         __syncthreads();
//         if (thid < d) {
//             int ai = offset*thid-1;
//             int bi = offset*(thid+1)-1;
//             temp[bi] += temp[ai];
//         }
//         offset *= 2;
//     }

//     if (thid == 0) { temp[length - 1] = 0; } // clear the last element

//     for (int d = 1; d < length; d *= 2) { // traverse down tree & build scan
//         offset >>= 1;
//         __syncthreads();
//         if (thid < d) {
//             int ai = offset*thid-1;
//             int bi = offset*(thid+1)-1;
//             int t   = temp[ai];
//             temp[ai] = temp[bi];
//             temp[bi] += t;
//         }
//     }
//     __syncthreads();

//     if (thid < length) {
//         output[thid] = temp[thid]; // write results to device memory
//     }
// }

__global__ 
void compute_cdf(int *histogram, int *cdf, int length) {
    int sum = 0;
    for (int i = 0; i < length; i++) {
        sum += histogram[i];
        cdf[i] = sum;
    }
}

__global__ 
void normalize_cdf(int *cdf, int size, int min_cdf) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i <= MAX_INTENSITY) {
        cdf[i] = ((cdf[i] - min_cdf) * MAX_INTENSITY) / (size - min_cdf);
    }
}

__global__ 
void equalize(png_byte *image, int *cdf, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        image[i] = cdf[image[i]];
    }
}


int main(int argc, char *argv[]) {
    // ... (load image into "image" array and initialize "histogram" and "cdf" arrays) ...
    
    Image img = {0};
    if (argc < 2) {
        printf("Usage: %s <image.png>\n", argv[0]);
        return 1;
    }

    char *input_file = argv[1];
    read_png_file(input_file, PNG_COLOR_TYPE_GRAY, &img);
    png_byte *image = img.data[0];
    int size = img.width * img.height;
    int histogram[MAX_INTENSITY + 1] = {0};
    int cdf[MAX_INTENSITY + 1] = {0};
    
    png_byte *d_image;
    int *d_cdf;
    int *d_histogram;
    CHECK(hipMalloc(&d_histogram, (MAX_INTENSITY + 1) * sizeof(int)));
    CHECK(hipMalloc(&d_image, size * sizeof(png_byte)));
    CHECK(hipMalloc(&d_cdf, (MAX_INTENSITY + 1) * sizeof(int)));

    CHECK(hipMemcpy(d_histogram, histogram, (MAX_INTENSITY + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_image, image, size * sizeof(png_byte), hipMemcpyHostToDevice));

    calculate_histogram<<<(size + 255) / 256, 256>>>(d_histogram, d_image, size);

    CHECK(hipMemcpy(histogram, d_histogram, (MAX_INTENSITY + 1) * sizeof(int), hipMemcpyDeviceToHost));

    int threadsPerBlock = 256;
    int blocksPerGrid = (MAX_INTENSITY + threadsPerBlock - 1) / threadsPerBlock;
    compute_cdf<<<blocksPerGrid, threadsPerBlock>>>(d_histogram, d_cdf, MAX_INTENSITY + 1);

    CHECK(hipMemcpy(cdf, d_cdf, (MAX_INTENSITY + 1) * sizeof(int), hipMemcpyDeviceToHost));

    int min_cdf = cdf[0]; // Assuming cdf[0] is the minimum value in the CDF
    normalize_cdf<<<(MAX_INTENSITY + 255) / 256, 256>>>(d_cdf, size, min_cdf);

    equalize<<<(size + 255) / 256, 256>>>(d_image, d_cdf, size);

    CHECK(hipMemcpy(image, d_image, size * sizeof(png_byte), hipMemcpyDeviceToHost));



    CHECK(hipFree(d_histogram));
    CHECK(hipFree(d_image));
    CHECK(hipFree(d_cdf));


    // ... (save image to disk) ...

    char filename[50];
    sprintf(filename, "equalizer%d.png");

    // Write the equalized image to a new file each time
    write_png_file(filename, &img);

    return 0;

    
}
