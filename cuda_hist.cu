#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <array>
#include <time.h>

#include <hip/hip_runtime.h>
#include "image.h"

#define MAX_INTENSITY 255

#define CHECK(call)                                                       \
{                                                                         \
   const hipError_t error = call;                                        \
   if (error != hipSuccess)                                              \
   {                                                                      \
      printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
      printf("code: %d, reason: %s\n", error, hipGetErrorString(error)); \
      exit(1);                                                            \
   }                                                                      \
}


__global__ 
void calculate_histogram(unsigned int *histogram, png_byte *image, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        atomicAdd(&histogram[image[i]], 1);
    }
}

__global__ 
void exclusive_scan(unsigned int *input, png_byte *output, int length) {
    extern __shared__ int temp[];  // allocated on invocation
    int thid = threadIdx.x;
    int offset = 1;

    temp[2*thid] = input[2*thid]; // load input into shared memory
    temp[2*thid+1] = input[2*thid+1];

    for (int d = length>>1; d > 0; d >>= 1) { // build sum in place up the tree
        __syncthreads();
        if (thid < d) {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    if (thid == 0) { temp[length - 1] = 0; } // clear the last element

    for (int d = 1; d < length; d *= 2) { // traverse down tree & build scan
        offset >>= 1;
        __syncthreads();
        if (thid < d) {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;
            int t   = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    output[2*thid] = temp[2*thid]; // write results to device memory
    output[2*thid+1] = temp[2*thid+1];
}

__global__ 
void normalize_cdf(png_byte *cdf, int size, int min_cdf) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i <= MAX_INTENSITY) {
        cdf[i] = ((cdf[i] - min_cdf) * MAX_INTENSITY) / (size - min_cdf);
    }
}

__global__ 
void equalize(png_byte *image, png_byte *cdf, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        image[i] = cdf[image[i]];
    }
}


int main(int argc, char *argv[]) {
    // ... (load image into "image" array and initialize "histogram" and "cdf" arrays) ...
    
    Image img = {0};
    if (argc < 2) {
        printf("Usage: %s <image.png>\n", argv[0]);
        return 1;
    }

    char *input_file = argv[1];
    read_png_file(input_file, PNG_COLOR_TYPE_GRAY, &img);
    png_byte *image = img.data[0];
    int size = img.width * img.height;
    int histogram[MAX_INTENSITY + 1] = {0};
    int cdf[MAX_INTENSITY + 1] = {0};
    
    png_byte *d_image, *d_cdf;
    unsigned int *d_histogram;
    CHECK(hipMalloc(&d_histogram, (MAX_INTENSITY + 1) * sizeof(int)));
    CHECK(hipMalloc(&d_image, size * sizeof(png_byte)));
    CHECK(hipMalloc(&d_cdf, (MAX_INTENSITY + 1) * sizeof(int)));

    CHECK(hipMemcpy(d_histogram, histogram, (MAX_INTENSITY + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_image, image, size * sizeof(png_byte), hipMemcpyHostToDevice));

    calculate_histogram<<<(size + 255) / 256, 256>>>(d_histogram, d_image, size);

    CHECK(hipMemcpy(histogram, d_histogram, (MAX_INTENSITY + 1) * sizeof(int), hipMemcpyDeviceToHost));

    exclusive_scan<<<1, MAX_INTENSITY + 1, (MAX_INTENSITY + 1) * sizeof(int)>>>(d_histogram, d_cdf, MAX_INTENSITY + 1);

    CHECK(hipMemcpy(cdf, d_cdf, (MAX_INTENSITY + 1) * sizeof(int), hipMemcpyDeviceToHost));

    int min_cdf = cdf[0]; // Assuming cdf[0] is the minimum value in the CDF
    normalize_cdf<<<(MAX_INTENSITY + 255) / 256, 256>>>(d_cdf, size, min_cdf);

    equalize<<<(size + 255) / 256, 256>>>(d_image, d_cdf, size);

    CHECK(hipMemcpy(image, d_image, size * sizeof(png_byte), hipMemcpyDeviceToHost));



    CHECK(hipFree(d_histogram));
    CHECK(hipFree(d_image));
    CHECK(hipFree(d_cdf));


    // ... (save image to disk) ...

    char filename[50];
    sprintf(filename, "equalizer%d.png");

    // Write the equalized image to a new file each time
    write_png_file(filename, &img);

    return 0;

    
}
